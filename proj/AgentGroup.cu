#include "hip/hip_runtime.h"
#include "AgentGroup.cuh"
#include "AgentsKernel.cuh"
#include "Map.cuh"

#include <math_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <>

#include <>

extern texture<float4, hipTextureType1D, hipReadModeElementType> g_previousAgentsPositions;
extern texture<int4, hipTextureType1D, hipReadModeElementType> g_neighborsData;
extern texture<int2, hipTextureType1D, hipReadModeElementType> g_agentsHashes;

__host__  AgentGroup::AgentGroup(int agentsCount) :
	_agentsCount(agentsCount)
{
	hipError_t status;
	status = hipHostMalloc(&_agents, sizeof(Agent)*_agentsCount);
	int sqrtAgentsCount = sqrt(static_cast<float>(_agentsCount));
	unsigned int actualAgentsCount = 0;

	float4* tempAgentsPosVel = new float4[agentsCount];

	for (int x = 0;;x++)
	{
		for (int y = 0;y<sqrtAgentsCount;y++)	
		{
			Agent agent;
			agent.Position(make_float2(x*20+3, y*20+3));
//			agent.Id = actualAgentsCount;
			agent.color = actualAgentsCount < agentsCount/2 ? make_float3(0.5f,0,0):make_float3(0,0.5f,0);
			agent.target = make_float2(actualAgentsCount < agentsCount/2 ? Map::WorldWidth() : 0,rand()/(32767/Map::WorldHeight())-1);
			tempAgentsPosVel[actualAgentsCount] = make_float4(agent.Position().x, agent.Position().y, agent.Velocity().x, agent.Velocity().y);
			_agents[actualAgentsCount++] = agent;
			if(actualAgentsCount>=_agentsCount)
				break;
		}
		if(actualAgentsCount>=_agentsCount)
			break;
	}

	status = hipMalloc(&_d_agents, sizeof(Agent)*agentsCount);
	status = hipMemcpy(_d_agents, _agents, sizeof(Agent)*agentsCount, hipMemcpyHostToDevice);

	status = hipMalloc(&_agentPositionVelocity, sizeof(float4)*agentsCount);
	status = hipMemcpy(_agentPositionVelocity, tempAgentsPosVel, sizeof(float4)*agentsCount, hipMemcpyHostToDevice);
	hipChannelFormatDesc agentsTextureDesc = hipCreateChannelDesc<float4>();
	status = hipMallocArray(&_previousAgentsArray, 
		&agentsTextureDesc,
		_agentsCount,
		0);
	status = hipMemcpyToArray(_previousAgentsArray, 0, 0, _agentPositionVelocity, sizeof(float4)*_agentsCount, hipMemcpyDeviceToDevice);
	//g_obstaclesTexture.normalized = false;
	status = hipBindTextureToArray(&g_previousAgentsPositions, _previousAgentsArray, &agentsTextureDesc);

	thrust::host_vector<int2> tempAgentsHashes = thrust::host_vector<int2>();
	for (int i = 0; i<_agentsCount;i++)
	{
		tempAgentsHashes.push_back(make_int2(i, 0));
	}
	_agentsHashes = tempAgentsHashes;
	hipChannelFormatDesc hashesTextureDesc = hipCreateChannelDesc<int2>();
	status = hipMallocArray(&_agentsHashesArray, 
		&hashesTextureDesc,
		_agentsCount,
		0);
	status = hipMemcpyToArray(_agentsHashesArray, 0, 0, thrust::raw_pointer_cast(&_agentsHashes[0]), sizeof(int2)*_agentsCount, hipMemcpyDeviceToDevice);
	g_agentsHashes.normalized = false;
	status = hipBindTextureToArray(&g_agentsHashes, _agentsHashesArray, &hashesTextureDesc);


	thrust::host_vector<int4> tempNeighborsData = thrust::host_vector<int4>();
	for (int i=0;i<Map::WidthNodesCount()*Map::HeightNodesCount();i++)
	{
		tempNeighborsData.push_back(make_int4(i, 0, 0, 0));
	}
	_neighborsData = tempNeighborsData;
	hipChannelFormatDesc neighborsTextureDesc = hipCreateChannelDesc<int4>();
	status = hipMallocArray(&_neighborsDataArray, 
		&neighborsTextureDesc,
		_neighborsData.size(),
		0);
	status = hipMemcpyToArray(_neighborsDataArray, 0, 0, thrust::raw_pointer_cast(&_neighborsData[0]), sizeof(int4)*_neighborsData.size(), hipMemcpyDeviceToDevice);
	g_neighborsData.normalized = false;
	status = hipBindTextureToArray(&g_neighborsData, _neighborsDataArray, &neighborsTextureDesc);
	//hipMalloc((void**)&_d_agents, sizeof(Agent)*agentsCount);
	//hipMemcpy(_d_agents, tempAgents, sizeof(Agent)*agentsCount, hipMemcpyHostToDevice);

	//delete [] tempAgents;
}

__host__  AgentGroup::~AgentGroup(void)
{
	//delete [] _agents;
}

struct CompareAgentHash
{
	__host__ __device__
	bool operator()(int2 a, int2 b)
	{
		return a.y < b.y;
	}
};
__host__ __device__ int GetNodeX(int x, int neighborsDataDim)
{
	return x/neighborsDataDim;
}
__host__ __device__ int GetNodeY(int y, int neighborsDataDim)
{
	return y/neighborsDataDim;
}

__host__ __device__ int GetNodeId( int x, int y, int neighborsDataDim )
{
	int xf = GetNodeX(x, neighborsDataDim);
	int yf = GetNodeY(y, neighborsDataDim);
	int result = xf+yf*neighborsDataDim;
	return result;
}
struct ComputeAgentHash
{
	//Agent* _agents;
	int _neighborsDataDim;
	ComputeAgentHash(/*Agent* agents, */int neighborsDataDim)
	{
		//_agents = agents;
		_neighborsDataDim = neighborsDataDim;
	}
	__device__
	int2 operator()(const int2& item)
	{
		float4 agent = tex1D(g_previousAgentsPositions, /*_agents[*/item.x);
		return make_int2(item.x, GetNodeId(agent.x, agent.y, _neighborsDataDim));
	}
};
struct ComputeNeighborsData
{
	int2* _agentsHashes;
	int _agentsCount;
	ComputeNeighborsData(int2* agentsHashes, int agentCount)
	{
		_agentsHashes = agentsHashes;
		_agentsCount = agentCount;
	}
	__host__ __device__
	int4 operator()(const int4& item)
	{
		int agentHashCounter = 0;
		while(agentHashCounter<_agentsCount && _agentsHashes[agentHashCounter].y!=item.x)
		{
			agentHashCounter++;
		}
		int4 result = make_int4(item.x, agentHashCounter, 0, 0);
		while(agentHashCounter<_agentsCount && _agentsHashes[agentHashCounter++].y==item.x)
		{
			result.z++;
		}
		return result;
	}
};

__global__ void UpdateAgentsKernel(//int3* neighborsData,
	int neighborsDataCount,
	int neighborsDataDim,
	/*int2* agentsHashes,*/
	int agentHashesCount,
	Agent* agents,
	//Agent* prevAgents,
	float4* agentsPositionVelocity,
	float4 boundary,
	float elapsedTime)
{
	int agentIndex = blockIdx.x*blockDim.x + threadIdx.x;
	Agent agent = agents[agentIndex];
		int nodeId = GetNodeId(agent.Position().x, agent.Position().y, neighborsDataDim);
		int neighborsCount = 0;
		int neighborsDataIndices[] = {
			nodeId-neighborsDataDim-1,
			nodeId-neighborsDataDim,
			nodeId-neighborsDataDim+1,
			nodeId- 1,
			nodeId + 0,
			nodeId + 1,
			nodeId + neighborsDataDim-1,
			nodeId + neighborsDataDim,
			nodeId + neighborsDataDim+1,
		};
		for (int i=0;i<9;i++)
		{
			int index = neighborsDataIndices[i];
			if(index < 0 || index >= neighborsDataCount)
				continue;
			neighborsCount += tex1D(g_neighborsData, index).z;
		}
		int* neighbors = new int[neighborsCount];
		int neighborsCounter = 0;
		for (int j = 0;j<9;j++)
		{
			int index = neighborsDataIndices[j];
			if(index<0||index>=neighborsDataCount)
				continue;
			int4 nodeData = tex1D(g_neighborsData, index);
			for (int i = nodeData.y; i<nodeData.y+nodeData.z;i++)
			{
				neighbors[neighborsCounter++] =  tex1D(g_agentsHashes, i).x;
			}
		}
		agent.Update(elapsedTime, neighbors, neighborsCount, boundary);
		delete [] neighbors;
		//syncthreads();
		agents[agentIndex] = agent;
		agentsPositionVelocity[agentIndex] = make_float4(agent.Position().x, agent.Position().y, agent.Velocity().x, agent.Velocity().y);
}

__global__ void GetAgentsPositionVelocity(float4* out)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	out[index] = tex1D(g_previousAgentsPositions, index);
}

__host__ void AgentGroup::Update( float elapsedTime )
{
	thrust::transform(_agentsHashes.begin(),_agentsHashes.end(), _agentsHashes.begin(), 
		ComputeAgentHash(sqrt((float)_neighborsData.size())));
	hipError_t status = hipDeviceSynchronize();
	thrust::sort(_agentsHashes.begin(),_agentsHashes.end(), CompareAgentHash());
	status = hipDeviceSynchronize();
	status = hipMemcpyToArray(_agentsHashesArray, 0, 0, thrust::raw_pointer_cast(&_agentsHashes[0]), sizeof(int2)*_agentsCount, hipMemcpyDeviceToDevice);
	thrust::transform(_neighborsData.begin(), _neighborsData.end(), _neighborsData.begin(), ComputeNeighborsData(
		(int2*)thrust::raw_pointer_cast(&_agentsHashes[0]), _agentsCount));
	status = hipDeviceSynchronize();
	status = hipMemcpyToArray(_neighborsDataArray, 0, 0, thrust::raw_pointer_cast(&_neighborsData[0]), sizeof(int4)*_neighborsData.size(), hipMemcpyDeviceToDevice);

	/*int3* neighborsData,
	int neighborsDataCount,
	int neighborsDataDim,
	int2* agentsHashes,
	int agentHashesCount,
	Agent* agents,
	Agent* prevAgents,
	float elapsedTime*/
	UpdateAgentsKernel<<<_agentsCount/128, 128>>>(
//		thrust::raw_pointer_cast(&_neighborsData[0]),
		_neighborsData.size(),
		sqrt(static_cast<float>(_neighborsData.size())),
		//thrust::raw_pointer_cast(&_agentsHashes[0]),
		_agentsCount,
		_d_agents,
		//d_prevAgents,
		_agentPositionVelocity,
		make_float4(0,0,Map::WorldWidth(),Map::WorldHeight()),
		elapsedTime);
	status = hipDeviceSynchronize();
	//status = hipFree(d_prevAgents);
	//status = hipFreeArray(previousAgentsArray);
	//size_t agentSize = sizeof(Agent);
	status = hipMemcpyToArray(_previousAgentsArray, 0, 0, _agentPositionVelocity, sizeof(float4)*_agentsCount, hipMemcpyDeviceToDevice);
	
	//status = hipBindTextureToArray(g_previousAgentsPositions, _previousAgentsArray);
	status = hipMemcpy(_agents, _d_agents, sizeof(Agent)*_agentsCount, hipMemcpyDeviceToHost);

	//float4* d_posVelCheck = NULL;
	//status = hipMalloc(&d_posVelCheck, sizeof(float4)*_agentsCount);
	//GetAgentsPositionVelocity<<<_agentsCount/1024,1024>>>(d_posVelCheck);
	//float4* posVelCheck = static_cast<float4*>(malloc(sizeof(float4)*_agentsCount));
	//status = hipMemcpy(posVelCheck, d_posVelCheck, sizeof(float4)*_agentsCount, hipMemcpyDeviceToHost);
}